#include "hip/hip_runtime.h"
#include "../HeaderFiles/Parallel2.h"
#include <sys/time.h>
#include <hip/hip_runtime.h>



__global__ void parMultiplyMatrixesCuda(int rowsA, int colsA, int rowsB, int colsB, double *arrA, double *arrB, double * arrC)
{
    double singleAcum;

            for(int memberNo = 0; memberNo < colsA; memberNo++){
                int elementNoA = blockIdx.x * colsA + memberNo;
                int elementNoB = threadIdx.x * rowsB + memberNo;
                singleAcum += arrA[elementNoA] * arrB[elementNoB];
            }
            arrC[blockIdx.x*colsB + threadIdx.x] = singleAcum;
    return;
}



extern "C" void runParallel2(int rowsA, int colsA, int rowsB, int colsB, double *arrA, double *arrB, double *arrC)
{   
    int blockSize = rowsA;
    int amountThreads = colsB;

    double * d_arrA;
    double * d_arrB;
    double * d_arrC;

    hipMalloc((void**)& d_arrA, rowsA * colsA * sizeof(double));
    hipMalloc((void**)& d_arrB, rowsB * colsB * sizeof(double));
    hipMalloc((void**)& d_arrC, rowsA * colsB * sizeof(double));

    hipMemcpy(d_arrA, arrA, rowsA * colsA * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_arrB, arrB, rowsB * colsB * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_arrC, arrC, rowsA * colsB * sizeof(double), hipMemcpyHostToDevice);


    struct timeval now, finish; 
    //long acum; 
       for (int i = 0; i < 5; i++){
            gettimeofday(&now, 0);
                
            //printf("Calling %d blocks and %d threads\n", blockSize, amountThreads);

            parMultiplyMatrixesCuda<<<blockSize, amountThreads>>>(rowsA, colsA, rowsB, colsB , d_arrA, d_arrB, d_arrC);
            hipDeviceSynchronize();
            gettimeofday(&finish, 0);
            long seconds = finish.tv_sec - now.tv_sec;
            long microseconds = finish.tv_usec - now.tv_usec;
            double elapsed = seconds + microseconds*1e-6;
            printf("Time measured: %.9f seconds.\n", elapsed);
            hipError_t error = hipGetLastError();
       }

    hipMemcpy(arrA, d_arrA, rowsA * colsA * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(arrB, d_arrB, rowsA * colsA * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(arrC, d_arrC, rowsA * colsB * sizeof(double), hipMemcpyDeviceToHost);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        printf("Cuda error: %s\n", hipGetErrorString(error));
    }
  
    return;
}



